#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;

#define M 7
#define N 7

// 矩阵加法 Kernel
__global__ void addMatrices(int *A, int *B, int *C, int m, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    if (idx < m && idy < n) {
        int index = idx * n + idy;
        C[index] = A[index] + B[index];
    }
}

// 初始化矩阵数据
void matrixInit(int* mat, int m, int n) {
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            mat[i*n + j] = i*n+j;
        }
    }
}

void printMatrix(int *mat, int m, int n) {
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            cout << mat[i*n + j] << "\t";
        }
        cout << endl;
    }
}

int main() {
    int size = M*N*sizeof(int);

    // 分配host内存
    int *host_A = (int *) malloc(size);
    int *host_B = (int *) malloc(size);
    int *host_C = (int *) malloc(size);

    // 初始化矩阵
    matrixInit(host_A, M, N);
    matrixInit(host_B, M, N);

    // 打印初始矩阵
    cout<<"矩阵A:"<<endl;
    printMatrix(host_A, M, N);
    cout<<"矩阵B:"<<endl;
    printMatrix(host_B, M, N);

    // 分配device内存
    int *dev_A, *dev_B, *dev_C;
    hipMalloc((void **)&dev_A, size);
    hipMalloc((void **)&dev_B, size);
    hipMalloc((void **)&dev_C, size);

    // 复制内存到device
    hipMemcpy((void *)dev_A, (void *)host_A, size, hipMemcpyHostToDevice);
    hipMemcpy((void *)dev_B, (void *)host_B, size, hipMemcpyHostToDevice);

    // 设置grid和block
    dim3 grid((M+1)/2, (N+1)/2);
    dim3 block(2, 2);

    // Launch the kernel
    addMatrices<<<grid, block>>>(dev_A, dev_B, dev_C, M, N);

    // 复制结果回host 
    hipMemcpy((void *)host_C, (void *)dev_C, size, hipMemcpyDeviceToHost);

    // 打印结果
    cout<<"矩阵加法结果:"<<endl;
    printMatrix(host_C, M, N);

    // 释放内存
    free(host_A);
    free(host_B);
    free(host_C);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    return 0;
}