#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;

const int M = 8;
const int N = 8;

__global__ void matrix_multiply(int *A, int *B, int *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        int sum = 0;
        for (int k = 0; k < width; k++) {
            sum += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = sum;
    }
}

int main() {
    int size = M * N * sizeof(int);

    int *host_A = (int *)malloc(size);
    int *host_B = (int *)malloc(size);
    int *host_C = (int *)malloc(size);

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            host_A[i * N + j] = i * N + j;
            host_B[i * N + j] = i * N + j;
            host_C[i * N + j] = 0;
        }
    }

    // 打印初始矩阵
    cout << "矩阵A:" << endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            cout << host_A[i * N + j] << " ";
        }
        cout << endl;
    }

    cout << "矩阵B:" << endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            cout << host_B[i * N + j] << " ";
        }
        cout << endl;
    }

    // 分配device内存
    int *dev_A, *dev_B, *dev_C;
    hipMalloc((void **)&dev_A, size);
    hipMalloc((void **)&dev_B, size);
    hipMalloc((void **)&dev_C, size);

    // 复制内存到device
    hipMemcpy(dev_A, host_A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, size, hipMemcpyHostToDevice);

    // 设置grid和block
    dim3 grid(M / 2, N / 2);
    dim3 block(2, 2);

    matrix_multiply<<<grid, block>>>(dev_A, dev_B, dev_C, N);

    hipMemcpy(host_C, dev_C, size, hipMemcpyDeviceToHost);

    cout << "乘法结果:" << endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            cout << host_C[i * N + j] << " ";
        }
        cout << endl;
    }

    // 释放内存
    free(host_A);
    free(host_B);
    free(host_C);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    return 0;
}